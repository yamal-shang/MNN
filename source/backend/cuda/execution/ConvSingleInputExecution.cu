#include "hip/hip_runtime.h"
//
//  ConvSingleInputExecution.cpp
//  MNN
//
//  Created by MNN on 2020/08/22.
//  Copyright © 2018, Alibaba Group Holding Limited
//

#include "ConvSingleInputExecution.hpp"

namespace MNN {
namespace CUDA {

__global__ void Im2Col(const ConvolutionCommon::Im2ColParameter* param,
        const MatMulParam* matmulParam,
        const float* A,
        __half* AP) {
    int eAlign = matmulParam->elhPack[0] * MATMULPACK;
    int lAlign = matmulParam->elhPack[1] * MATMULPACK;
    int maxCount = eAlign * lAlign;
    int kernelCount = param->kernelX * param->kernelY;
    for (size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < maxCount; index += blockDim.x * gridDim.x) {
        int eIndex = index % eAlign;
        int lIndex = index / eAlign;
        // Compute for dest
        int eU = eIndex / MATMULPACK;
        int eR = eIndex % MATMULPACK;
        int lU = lIndex / MATMULPACK;
        int lR = lIndex % MATMULPACK;
        auto dstOffset = eU * matmulParam->elhPack[1] * (MATMULPACK * MATMULPACK) + lU * (MATMULPACK * MATMULPACK) + eR * MATMULPACK + lR;
        if (eIndex >= matmulParam->elh[0] || lIndex >= matmulParam->elh[1]) {
            AP[dstOffset] = 0.0;
            continue;
        }
        // Compute for source
        int ox = eIndex % param->ow;
        int oy = eIndex / param->ow;
        int ob = oy / param->oh;
        oy = oy % param->oh;
        int sz = lIndex / kernelCount;
        int kI = lIndex % kernelCount;
        int ksx = kI % param->kernelX;
        int ksy = kI / param->kernelX;

        int sx = ox * param->strideX + ksx * param->dilateX - param->padX;
        int sy = oy * param->strideY + ksy * param->dilateY - param->padY;
        if (sx >= 0 && sx < param->iw) {
            if (sy >=0 && sy < param->ih) {
                __half value = A[sz * param->ih * param->iw + ob * param->iw * param->ih * param->icDiv4 + sy * param->iw + sx];
                AP[dstOffset] = value;
                continue;
            }
        }
        AP[dstOffset] = 0.0;
    }
}


ConvSingleInputExecution::Resource::Resource(Backend* bn, const MNN::Op* op) {
    mBackend = bn;
    auto runtime = static_cast<CUDABackend*>(bn)->getCUDARuntime();

    auto conv       = op->main_as_Convolution2D();
    auto common     = conv->common();
    mKernelInfo.kernelX        = common->kernelX();
    mKernelInfo.kernelY        = common->kernelY();
    mKernelInfo.groups         = common->group();
    mKernelInfo.strideX        = common->strideX();
    mKernelInfo.strideY        = common->strideY();
    mKernelInfo.dilateX        = common->dilateX();
    mKernelInfo.dilateY        = common->dilateY();
    mKernelInfo.activationType = common->relu() ? 1 : (common->relu6() ? 2 : 0);

    //weight host->device
    const float* filterDataPtr = nullptr;
    int weightSize = 0;
    std::shared_ptr<ConvolutionCommon::Int8Common> quanCommon;
    ConvolutionCommon::getConvParameters(&quanCommon, conv, &filterDataPtr, &weightSize);
    mKernelInfo.kernelN = common->outputCount();
    mKernelInfo.kernelC = weightSize / mKernelInfo.kernelN / mKernelInfo.kernelX / mKernelInfo.kernelY;

    MatMulParam param;
    int e = 0;
    int l = mKernelInfo.kernelX * mKernelInfo.kernelY * mKernelInfo.kernelC;
    int h = mKernelInfo.kernelN;
    param.elh[0] = e;
    param.elh[1] = l;
    param.elh[2] = h;
    param.elhPack[0] = UP_DIV(e, 16);
    param.elhPack[1] = UP_DIV(l, 16);
    param.elhPack[2] = UP_DIV(h, 16);
    param.bStride[0] = 0;
    param.bStride[1] = 1;
    param.bStride[2] = l;

    auto gpuParam = static_cast<CUDABackend*>(bn)->getStaticBufferPool()->alloc(sizeof(MatMulParam));
    auto tempCacheBuffer = static_cast<CUDABackend*>(bn)->getStaticBufferPool()->alloc(weightSize * sizeof(float));
    float* cacheWeight = (float*)((uint8_t*)tempCacheBuffer.first + tempCacheBuffer.second);
    runtime->memcpy(cacheWeight, filterDataPtr, weightSize * sizeof(float), MNNMemcpyHostToDevice);
    runtime->memcpy((uint8_t*)gpuParam.first + gpuParam.second, &param, sizeof(MatMulParam), MNNMemcpyHostToDevice);
    // Reorder weight
    weightTensor.reset(Tensor::createDevice<int16_t>({param.elhPack[1] * param.elhPack[2] * (MATMULPACK * MATMULPACK)}));
    bn->onAcquireBuffer(weightTensor.get(), Backend::STATIC);
    mFilter = (void *)weightTensor.get()->buffer().device;
    GemmPrepareRerange(runtime, &param, (const MatMulParam*)((uint8_t*)gpuParam.first + gpuParam.second), nullptr, nullptr, cacheWeight, (__half*)mFilter);
    static_cast<CUDABackend*>(bn)->getStaticBufferPool()->free(tempCacheBuffer);
    static_cast<CUDABackend*>(bn)->getStaticBufferPool()->free(gpuParam);

    // Copy Bias
    int biasSize = conv->bias()->size();
    biasTensor.reset(Tensor::createDevice<float>({biasSize}));
    bn->onAcquireBuffer(biasTensor.get(), Backend::STATIC);
    mBias = (void *)biasTensor.get()->buffer().device;
    cuda_check(hipMemcpy(mBias, conv->bias()->data(), conv->bias()->size()*sizeof(float), hipMemcpyHostToDevice));
}

ConvSingleInputExecution::Resource::~Resource() {
    // Do nothing
}
ConvSingleInputExecution::ConvSingleInputExecution(Backend* backend, const MNN::Op* op, std::shared_ptr<Resource> res) : Execution(backend), mOp(op) {
    mResource = res;
    auto runtime = static_cast<CUDABackend*>(backend)->getCUDARuntime();
    auto staticPool = static_cast<CUDABackend*>(backend)->getStaticBufferPool();
    mGpuMatMulParam = staticPool->alloc(sizeof(MatMulParam));
    mGpuIm2ColParam = staticPool->alloc(sizeof(ConvolutionCommon::Im2ColParameter));
}

ConvSingleInputExecution::~ConvSingleInputExecution() {
    auto staticPool = static_cast<CUDABackend*>(backend())->getStaticBufferPool();
    staticPool->free(mGpuMatMulParam);
    staticPool->free(mGpuIm2ColParam);
}
bool ConvSingleInputExecution::onClone(Backend* bn, const Op* op, Execution** dst) {
    if (!mValid) {
        return false;
    }
    if (nullptr == dst) {
        return true;
    }
    auto dstExe = new ConvSingleInputExecution(bn, op, mResource);
    *dst = dstExe;
    return true;
}


ErrorCode ConvSingleInputExecution::onResize(const std::vector<Tensor*> &inputs, const std::vector<Tensor*> &outputs) {
    auto runtime = static_cast<CUDABackend*>(backend())->getCUDARuntime();
    auto input = inputs[0], output = outputs[0];
    const int UNIT = 1;
    auto convCommon = mOp->main_as_Convolution2D()->common();
    auto pads = ConvolutionCommon::convolutionPadFull(input, output, mOp->main_as_Convolution2D()->common());
    mIm2ColParamter.dilateX         = convCommon->dilateX();
    mIm2ColParamter.dilateY         = convCommon->dilateY();
    mIm2ColParamter.strideX         = convCommon->strideX();
    mIm2ColParamter.strideY         = convCommon->strideY();
    mIm2ColParamter.icDiv4          = input->channel();
    mIm2ColParamter.kernelX         = convCommon->kernelX();
    mIm2ColParamter.kernelY         = convCommon->kernelY();
    mIm2ColParamter.padX = std::get<0>(pads);
    mIm2ColParamter.padY = std::get<1>(pads);

    mIm2ColParamter.ih = input->height();
    mIm2ColParamter.iw = input->width();
    mIm2ColParamter.oh = output->height();
    mIm2ColParamter.ow = output->width();
    mIm2ColParamter.srcZStep = input->height() * input->width() * UNIT * input->batch();
    mIm2ColParamter.srcYStep = input->width() * UNIT;
    mIm2ColParamter.packCUnit = UNIT;

    runtime->memcpy((uint8_t*)mGpuIm2ColParam.first + mGpuIm2ColParam.second, &mIm2ColParamter, sizeof(ConvolutionCommon::Im2ColParameter), MNNMemcpyHostToDevice);

    int e = output->height() * output->width() * output->batch();
    int l = input->channel() * mIm2ColParamter.kernelX * mIm2ColParamter.kernelY;
    int h = output->channel();
    mMatMulParam.elh[0] = e;
    mMatMulParam.elh[1] = l;
    mMatMulParam.elh[2] = h;
    mMatMulParam.elhPack[0] = UP_DIV(e, 16);
    mMatMulParam.elhPack[1] = UP_DIV(l, 16);
    mMatMulParam.elhPack[2] = UP_DIV(h, 16);
    mMatMulParam.cStride[0] = mIm2ColParamter.ow * mIm2ColParamter.oh * h;
    mMatMulParam.cStride[1] = 1;
    mMatMulParam.cStride[2] = mIm2ColParamter.ow * mIm2ColParamter.oh;
    mMatMulParam.split[0] = 1;
    mMatMulParam.split[1] = 1;
    mMatMulParam.split[2] = mIm2ColParamter.ow * mIm2ColParamter.oh;
    if (convCommon->relu()) {
        mMatMulParam.minValue = 0.0f;
    }
    if (convCommon->relu6()) {
        mMatMulParam.minValue = 0.0f;
        mMatMulParam.maxValue = 6.0f;
    }
    runtime->memcpy((uint8_t*)mGpuMatMulParam.first + mGpuMatMulParam.second, &mMatMulParam, sizeof(MatMulParam), MNNMemcpyHostToDevice);

    auto pool = static_cast<CUDABackend*>(backend())->getBufferPool();
    auto buffer = pool->alloc(sizeof(__half) * mMatMulParam.elhPack[0] * mMatMulParam.elhPack[1] * MATMULPACK * MATMULPACK);
    mIm2ColBuffer = (__half*)((uint8_t*)buffer.first + buffer.second);
    pool->free(buffer);
    return NO_ERROR;
}

ErrorCode ConvSingleInputExecution::onExecute(const std::vector<Tensor*> &inputs, const std::vector<Tensor*> &outputs) {
    //MNN_PRINT("cuda convSingleInput onExecute in, inputsize:%d %d\n", (int)inputs.size(), workspace_size_);
    MNN_ASSERT(inputs.size() == 1);
    MNN_ASSERT(outputs.size() == 1);

    auto runtime = static_cast<CUDABackend*>(backend())->getCUDARuntime();
    const void *input_addr = (const void*)inputs[0]->deviceId();
    const void *filter_addr = mResource->mFilter;
    const void *bias_addr = mResource->mBias;

    void *output_addr = (void*)outputs[0]->deviceId();
    auto& prop = runtime->prop();
    int threads_num = prop.maxThreadsPerBlock;
    int cores = prop.multiProcessorCount;
    auto gpuIm2Col = (const ConvolutionCommon::Im2ColParameter*)((uint8_t*)mGpuIm2ColParam.first + mGpuIm2ColParam.second);
    auto gpuMatMul = (const MatMulParam*)((uint8_t*)mGpuMatMulParam.first + mGpuMatMulParam.second);
    //runtime->memset(mIm2ColBuffer, 0, mMatMulParam.elhPack[0] * mMatMulParam.elhPack[1] * sizeof(__half) * (MATMULPACK * MATMULPACK));
    Im2Col<<<cores, threads_num>>>(gpuIm2Col, gpuMatMul, (const float*)input_addr, mIm2ColBuffer);
    GemmPackedMain(runtime, &mMatMulParam, gpuMatMul, (float*)output_addr, (const __half*)mIm2ColBuffer, (const __half*)filter_addr, (const float*)bias_addr);

    return NO_ERROR;
}

class CUDAConvolutionCreator : public CUDABackend::Creator {
public:
    virtual Execution* onCreate(const std::vector<Tensor*>& inputs, const std::vector<Tensor*>& outputs, 
            const MNN::Op* op, Backend* backend) const override {
        if (nullptr != op->main_as_Convolution2D()->quanParameter()) {
            auto quan = op->main_as_Convolution2D()->quanParameter();
            if (1 == quan->type() || 2 == quan->type()) {
                if (quan->has_scaleInt()) {
                    // Don't support IDST-int8 because of error
                    return nullptr;
                }
            }
        }
        std::shared_ptr<ConvSingleInputExecution::Resource> resource(new ConvSingleInputExecution::Resource(backend, op));
        return new ConvSingleInputExecution(backend, op, resource);
    }
};

CUDACreatorRegister<CUDAConvolutionCreator> __ConvExecution(OpType_Convolution);

}// namespace CUDA
}// namespace MNN
